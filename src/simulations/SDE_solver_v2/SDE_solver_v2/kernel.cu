#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
 
#include <hiprand/hiprand.h> 

#define CUDA_CALL(x) do { if((x)!=hipSuccess){ \
	printf("Error at %s:%d\n",__FILE__,__LINE__); \
	return EXIT_FAILURE;}} while(0) 

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__); \
	return EXIT_FAILURE;}} while(0)

using namespace std;



__global__ void SDE_simulations(double *x1, double *x2, double *temp1, double *temp2, int n, int time, float *d_Normal, float *d_Uni)
{
	int i = threadIdx.x;
	double a1 = 1.0;
	double a2 = 1.0;
	double b1 = 1.0;
	double b2 = 1.0;
	double k1 = 1.0;
	double k2 = 1.0;
	double H = 4.0;
	double S = 0.5;

	double dt = 10.0 / (double)time;


	if (i<n)
	{
		x1[i] = (double)d_Uni[2*i]*3.0;
		x2[i] = (double)d_Uni[2*i+1]*3.0;
		//x1[i] = 3.0;
		//x2[i] = 3.0;

		for (int j = 0; j < time - 1; j++)
		{
			temp1[i] = x1[i];
			temp2[i] = x2[i];

			x1[i] = temp1[i] + dt*(a1*pow(temp1[i], H) / (pow(temp1[i], H) + pow(S, H)) + b1*pow(S, H) / (pow(temp2[i], H) + pow(S, H)) - k1*temp1[i])+ sqrt(fabs(a1*pow(temp1[i], H) / (pow(temp1[i], H) + pow(S, H)) + b1*pow(S, H) / (pow(temp2[i], H) + pow(S, H)) - k1*temp1[i]))*(double)d_Normal[i*time + 2 * j];
			x2[i] = temp2[i] + dt*(a2*pow(temp2[i], H) / (pow(temp2[i], H) + pow(S, H)) + b2*pow(S, H) / (pow(temp1[i], H) + pow(S, H)) - k2*temp2[i])+ sqrt(fabs(a2*pow(temp2[i], H) / (pow(temp2[i], H) + pow(S, H)) + b2*pow(S, H) / (pow(temp1[i], H) + pow(S, H)) - k2*temp2[i]))*(double)d_Normal[i*time + 2 * j + 1];
		}
	}
}



int main()
{
	int n = 1000000;
	int time = 10;

	//CPU Memory variables
	double *x1, *x2 , *temp1, *temp2;
	x1 = (double *)malloc(n* sizeof(double));
	x2 = (double *)malloc(n* sizeof(double));
	temp1 = (double *)malloc(n * sizeof(double));
	temp2 = (double *)malloc(n * sizeof(double));

	//GPU Memory variables
	double *d_x2, *d_x1 , *d_temp1, *d_temp2;

	hipMalloc(&d_x1, n * sizeof(double));
	hipMalloc(&d_x2, n * sizeof(double));
	hipMalloc(&d_temp1, n * sizeof(double));
	hipMalloc(&d_temp2, n * sizeof(double));

	hiprandGenerator_t gen;
	float *d_Normal, *d_Uni;//, *hostData;
	//hostData = (double *)calloc(n, sizeof(double));

	CUDA_CALL(hipMalloc(&d_Normal, 2 * time*n * sizeof(float)));
	CUDA_CALL(hipMalloc(&d_Uni, 2*n * sizeof(float)));
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
	CURAND_CALL(hiprandGenerateNormal(gen, d_Normal, 2 * n*time, 0.0, 10.0/(float)time));
	CURAND_CALL(hiprandGenerateUniform(gen, d_Uni, 2 * n));
	

	/*for (int i = 0; i < n; i++)
	{
		x1[i] = 1.0;
		x2[i] = 1.0;
	}*/

	//cudaMemcpy(d_x1, x1, n* sizeof(double), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_x2, x2, n* sizeof(double), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_temp1, temp1, n*sizeof(double), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_temp2, temp2, n*sizeof(double), cudaMemcpyHostToDevice);

	SDE_simulations<<<1,512>>>(d_x1, d_x2, d_temp1, d_temp2, n, time, d_Normal, d_Uni);

	hipMemcpy(x1, d_x1, n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(x2, d_x2, n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp1, d_temp1, n * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp2, d_temp2, n * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
	{
		printf("%f - %f\n", x1[i], x2[i]);
	}


	CURAND_CALL(hiprandDestroyGenerator(gen));
	CUDA_CALL(hipFree(d_Normal));
	CUDA_CALL(hipFree(d_Uni));
	free(x1);
	free(x2);
	free(temp1);
	free(temp2);
	hipFree(d_x1);
	hipFree(d_x2);
	hipFree(d_temp1);
	hipFree(d_temp2);

	return 0;
}

